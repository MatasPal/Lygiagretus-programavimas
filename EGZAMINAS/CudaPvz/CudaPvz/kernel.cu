#include "hip/hip_runtime.h"
﻿#define __HIPCC__
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iomanip>
#include <fstream>
#include <iostream>

using namespace std;
__global__ void printNumbers(int* maxValue1)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < 100)
    {
        printf("%d\n", tid);

        atomicMax(maxValue1, tid);
    }
}

__global__ void printSquares(int* maxValue1)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < 100)
    {
        printf("%d\n", tid * tid);
    }

    atomicMax(maxValue1, tid * tid);
}

int main() {
    int* max_value;

    hipMalloc((void**)&max_value, sizeof(int));

    int initMax = -1;
    hipMemcpy(max_value, &initMax, sizeof(int), hipMemcpyHostToDevice);

    printNumbers << <2, 50 >> > (max_value);
    hipDeviceSynchronize();

    int maxVal0;
    hipMemcpy(&maxVal0, max_value, sizeof(int), hipMemcpyDeviceToHost);

    printf("Max value: %d\n", maxVal0);

    printSquares << <2, 50 >> > (max_value);
    hipDeviceSynchronize();

    int maxVal;
    hipMemcpy(&maxVal, max_value, sizeof(int), hipMemcpyDeviceToHost);

    printf("Max value: %d\n", maxVal);

    hipFree(max_value);

    return 0;
}
